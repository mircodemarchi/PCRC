#include "hip/hip_runtime.h"
/**
 * @file pcrc16.cu
 * @date 08/09/2020
 * @author Mirco De Marchi
 * @brief Source of 16 bit CRC parallel and sequential algorithms.
 */

#include "pcrc16.cuh"

#include <chrono>
#include <random>
#include <arpa/inet.h>

#include "Timer.cuh"
#include "CheckError.cuh"

#include "crc16-bitwise.h"
#include "crc16-bytewise.h"
#include "mod2.h"
#include "mul2.h"

using namespace timer;
//------------------------------------------------------------------------------

#define M 2             ///< Size of CRC result.
//------------------------------------------------------------------------------

/**
 * @brief Device CRC16 kernel executed by each GPU thread.
 * @param d_message     Message from which calculate the CRC value.
 * @param d_beta        Array of beta factor.
 * @param d_generator   Polynomial generator.
 * @param d_crc         Pointer to the result of the CRC value.
 */
__global__
static void pcrc16_kernel(const uint16_t* d_message, 
                          const uint16_t* d_beta,
                          const uint32_t d_generator,
                          uint16_t *d_partial_crc);

__global__
static void pcrc16_kernel_reduction(const uint16_t* d_message, 
                                    const uint16_t* d_beta,
                                    const uint32_t d_generator,
                                    uint16_t *d_partial_crc);                          

//------------------------------------------------------------------------------

void *pcrc16_init_common(const constants_t *c)
{
    const size_t N = c->N;

    // Host allocation.
    pcrc16_params_t *params = new pcrc16_params_t;
    params->h_message      = new uint8_t[N];
    params->h_message_16   = new uint16_t[N/M];
    params->h_beta         = new uint16_t[N/M];
    params->h_crc_partial_res_dev = new uint16_t[CEIL(N, BLOCK_SIZE)/M];

    params->generator = CRC16_CCITT;

    // Host initialization.
    params->h_crc_res_dev  = 0x0000;
    params->h_crc_res_host = 0x0000;

    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<uint8_t> distribution(0x00, 0xFF);

    for (size_t i = 0; i < N; i++) 
    {
        params->h_message[i] = distribution(generator);
    }

    // Convert h_message in h_message_16.
    memcpy(params->h_message_16, params->h_message, N);
    for (size_t i = 0; i < N/M; i++)
    {
        params->h_message_16[i] = ntohs(params->h_message_16[i]);
    }

    // Generate beta array.

    // Slow method.
    // for (size_t i = 0; i < N/M; i++) 
    // {
    //     size_t shift_buffer_length = M * (i + 1);
    //     uint8_t *shift_buffer = new uint8_t[shift_buffer_length + 1]();
    //     shift_buffer[0] = 0x01;
    //     params->h_beta[N/M - i - 1] = (uint16_t) mod2_32(
    //         shift_buffer, shift_buffer_length + 1, params->generator + 0x10000);
    //     delete[] shift_buffer;
    // }

    // Fast method.
    uint8_t *beta0 = new uint8_t[M + 1]();
    uint8_t mul32_arr[4] = {};
    beta0[0] = 0x01;
    params->h_beta[N/M - 1] = (uint16_t) mod2_32(beta0, M + 1, 
                              params->generator + 0x10000);
    delete[] beta0;
    for (size_t i = 1; i < N/M; i++) 
    {
        uint32_t mul32 = mul2_16(params->h_beta[N/M - i], params->h_beta[N/M - 1]);
        mul32 = htobe32(mul32);
        memcpy(mul32_arr, &mul32, 4);
        params->h_beta[N/M - i - 1] = (uint16_t) mod2_32(
            mul32_arr, 4, params->generator + 0x10000);
    }

    return (void *) params;
}

void *pcrc16_init_device(const constants_t *c, void *params)
{
    const size_t N = c->N;
    pcrc16_params_t *pcrc16_params = (pcrc16_params_t *) params;

    // Reset tmp data.
    pcrc16_params->h_crc_res_dev  = 0x00;
    pcrc16_params->h_crc_res_host = 0x00;

    // Device allocation.
    SAFE_CALL(hipMalloc(&pcrc16_params->d_message , sizeof(uint16_t) * N / M))
    SAFE_CALL(hipMalloc(&pcrc16_params->d_beta    , sizeof(uint16_t) * N / M))
    SAFE_CALL(hipMalloc(&pcrc16_params->d_crc_partial_res, 
                         sizeof(uint16_t) * CEIL(N, BLOCK_SIZE) / M))

    return params;
}

void *pcrc16_init_device_reduction(const constants_t *c, void *params)
{
    return pcrc16_init_device(c, params);
}

void *pcrc16_init_device_task_parallelism(const constants_t *c, void *params)
{
    const uint8_t STREAM_DIM = c->STREAM_DIM;
    const size_t  SEG_SIZE   = c->SEG_SIZE;
    pcrc16_params_t *pcrc16_params = (pcrc16_params_t *) params;

    // Reset tmp data.
    pcrc16_params->h_crc_res_dev  = 0x00;
    pcrc16_params->h_crc_res_host = 0x00;
    
    // Device allocation.
    SAFE_CALL(hipMalloc(&pcrc16_params->d_message, 
              sizeof(uint16_t) * SEG_SIZE * STREAM_DIM / M))
    SAFE_CALL(hipMalloc(&pcrc16_params->d_beta, 
              sizeof(uint16_t) * SEG_SIZE * STREAM_DIM / M))
    SAFE_CALL(hipMalloc(&pcrc16_params->d_crc_partial_res, 
              sizeof(uint16_t) * CEIL(SEG_SIZE, BLOCK_SIZE) * STREAM_DIM / M))

    return params;
}

void *pcrc16_init(const constants_t *c)
{
    return pcrc16_init_device(c, pcrc16_init_common(c));
}

void *pcrc16_init_reduction(const constants_t *c)
{
    return pcrc16_init_device_reduction(c, pcrc16_init_common(c));
}

void *pcrc16_init_task_parallelism(const constants_t *c)
{
    return pcrc16_init_device_task_parallelism(c, pcrc16_init_common(c));
}

void pcrc16_sequential(const constants_t *c, void *params, host_time_t *h_time)
{
    Timer<HOST> TM_host;

    const size_t N = c->N;

    uint8_t *message  = ((pcrc16_params_t *) params)->h_message;
    uint16_t generator = ((pcrc16_params_t *) params)->generator;
    // TODO: implement crc16 with generator.

    TM_host.start();
    uint16_t crc = crc16_bitwise(message, N);
    ((pcrc16_params_t *) params)->h_crc_res_host = crc;
    TM_host.stop();

    h_time->is_initialized = true;
    h_time->exec_time = TM_host.duration();
}

void pcrc16_sequential_bytewise(const constants_t *c, void *params, host_time_t *h_time)
{
    Timer<HOST> TM_host;

    const size_t N = c->N;

    uint8_t *message  = ((pcrc16_params_t *) params)->h_message;
    uint16_t generator = ((pcrc16_params_t *) params)->generator;
    // TODO: implement crc16 with generator.

    TM_host.start();
    uint16_t crc = crc16_bytewise(message, N, crc16_lu);
    ((pcrc16_params_t *) params)->h_crc_res_host = crc;
    TM_host.stop();

    h_time->is_initialized = true;
    h_time->exec_time = TM_host.duration();
}

void pcrc16_parallel(const constants_t *c, void *params, device_time_t *d_time)
{
    Timer<DEVICE> TM_device_kernel, TM_device_htod, TM_device_dtoh;
    Timer<HOST> TM_host;

    const size_t N = c->N;

    uint16_t *d_message = ((pcrc16_params_t *) params)->d_message;
    uint16_t *d_beta    = ((pcrc16_params_t *) params)->d_beta; 
    uint16_t *d_crc_partial_res = ((pcrc16_params_t *) params)->d_crc_partial_res;

    uint16_t *h_message = ((pcrc16_params_t *) params)->h_message_16;
    uint16_t *h_beta    = ((pcrc16_params_t *) params)->h_beta;
    uint16_t generator = ((pcrc16_params_t *) params)->generator;
    uint16_t *h_crc_partial_res_dev = 
        ((pcrc16_params_t *) params)->h_crc_partial_res_dev;

    // Device copy inputs.
    TM_device_htod.start();
    SAFE_CALL(hipMemcpy(d_message, h_message, sizeof(uint16_t) * N/M, 
        hipMemcpyHostToDevice))
    SAFE_CALL(hipMemcpy(d_beta, h_beta, sizeof(uint16_t) * N/M, 
        hipMemcpyHostToDevice))
    TM_device_htod.stop();

    TM_device_kernel.start();
    // Device dim.
    dim3 DimGrid((N/M) / BLOCK_SIZE, 1, 1);
    if ((N/M) % BLOCK_SIZE) DimGrid.x++;
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    // Device kernel call.
    pcrc16_kernel<<< DimGrid, DimBlock >>>(d_message, d_beta, 
        (uint32_t) generator + 0x10000, d_crc_partial_res);
    CHECK_CUDA_ERROR
    TM_device_kernel.stop();

    // Device copy result.
    TM_device_dtoh.start();
    SAFE_CALL(hipMemcpy(h_crc_partial_res_dev, d_crc_partial_res, 
        sizeof(uint16_t) * CEIL(N, BLOCK_SIZE) / M, hipMemcpyDeviceToHost))
    TM_device_dtoh.stop();
    
    TM_host.start();
    for (size_t i = 0; i < (CEIL(N, BLOCK_SIZE) / M); i++)
    {
        ((pcrc16_params_t *) params)->h_crc_res_dev ^= h_crc_partial_res_dev[i];
    }
    TM_host.stop();

    d_time->is_initialized      = true;
    d_time->is_task_parallelism = false;
    d_time->htod_time   = TM_device_htod.duration();
    d_time->kernel_time = TM_device_kernel.duration() + TM_host.duration();
    d_time->dtoh_time   = TM_device_dtoh.duration();
}

void pcrc16_parallel_reduction(const constants_t *c, void *params, device_time_t *d_time)
{
    Timer<DEVICE> TM_device_kernel, TM_device_htod, TM_device_dtoh;
    Timer<HOST> TM_host;

    const size_t N = c->N;

    uint16_t *d_message = ((pcrc16_params_t *) params)->d_message;
    uint16_t *d_beta    = ((pcrc16_params_t *) params)->d_beta; 
    uint16_t *d_crc_partial_res = ((pcrc16_params_t *) params)->d_crc_partial_res;

    uint16_t *h_message = ((pcrc16_params_t *) params)->h_message_16;
    uint16_t *h_beta    = ((pcrc16_params_t *) params)->h_beta;
    uint16_t generator = ((pcrc16_params_t *) params)->generator;
    uint16_t *h_crc_partial_res_dev = 
        ((pcrc16_params_t *) params)->h_crc_partial_res_dev;

    // Device copy inputs.
    TM_device_htod.start();
    SAFE_CALL(hipMemcpy(d_message, h_message, sizeof(uint16_t) * N/M, 
        hipMemcpyHostToDevice))
    SAFE_CALL(hipMemcpy(d_beta, h_beta, sizeof(uint16_t) * N/M, 
        hipMemcpyHostToDevice))
    TM_device_htod.stop();

    TM_device_kernel.start();
    // Device dim.
    dim3 DimGrid((N/M) / BLOCK_SIZE, 1, 1);
    if ((N/M) % BLOCK_SIZE) DimGrid.x++;
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    // Device kernel call.
    pcrc16_kernel_reduction<<< DimGrid, DimBlock >>>(d_message, d_beta, 
        (uint32_t) generator + 0x10000, d_crc_partial_res);
    CHECK_CUDA_ERROR
    TM_device_kernel.stop();

    // Device copy result.
    TM_device_dtoh.start();
    SAFE_CALL(hipMemcpy(h_crc_partial_res_dev, d_crc_partial_res, 
        sizeof(uint16_t) * CEIL(N, BLOCK_SIZE) / M, hipMemcpyDeviceToHost))
    TM_device_dtoh.stop();
    
    TM_host.start();
    for (size_t i = 0; i < (CEIL(N, BLOCK_SIZE) / M); i++) 
    {
        ((pcrc16_params_t *) params)->h_crc_res_dev ^= h_crc_partial_res_dev[i];
    }
    TM_host.stop();

    d_time->is_initialized      = true;
    d_time->is_task_parallelism = false;
    d_time->htod_time   = TM_device_htod.duration();
    d_time->kernel_time = TM_device_kernel.duration() + TM_host.duration();
    d_time->dtoh_time   = TM_device_dtoh.duration();
}

void pcrc16_parallel_task_parallelism(const constants_t *c, void *params, device_time_t *d_time)
{
    Timer<DEVICE> TM_device;
    Timer<HOST> TM_host;

    const size_t  N = c->N;
    const uint8_t STREAM_DIM = c->STREAM_DIM;
    const size_t  SEG_SIZE   = c->SEG_SIZE;

    uint16_t *d_message = ((pcrc16_params_t *) params)->d_message;
    uint16_t *d_beta    = ((pcrc16_params_t *) params)->d_beta; 
    uint16_t *d_crc_partial_res = ((pcrc16_params_t *) params)->d_crc_partial_res;

    uint16_t *h_message = ((pcrc16_params_t *) params)->h_message_16;
    uint16_t *h_beta    = ((pcrc16_params_t *) params)->h_beta;
    uint16_t generator = ((pcrc16_params_t *) params)->generator;
    uint16_t *h_crc_partial_res_dev = 
        ((pcrc16_params_t *) params)->h_crc_partial_res_dev;

    // TASK PARALLELISM
    TM_device.start();
    hipStream_t stream[STREAM_DIM];
    for (uint8_t i = 0; i < STREAM_DIM; i++) {
        hipStreamCreate(stream + i);
    }

    // Reminder: each STREAM takes one SEGMENT.
    for (int i = 0; i < N; i += SEG_SIZE * STREAM_DIM) {
        // 1. Copy inputs for every streams.
        for (int stream_index = 0; stream_index < STREAM_DIM; stream_index++) {
            int h_input_offset = i + (stream_index * SEG_SIZE);
            int d_input_offset = stream_index * SEG_SIZE;
            SAFE_CALL( 
                hipMemcpyAsync(
                    d_message + d_input_offset / M, 
                    h_message + h_input_offset / M,  
                    sizeof(uint16_t) * SEG_SIZE / M, 
                    hipMemcpyHostToDevice, 
                    stream[stream_index]) 
            )
            SAFE_CALL( 
                hipMemcpyAsync(
                    d_beta + d_input_offset / M, 
                    h_beta + h_input_offset / M, 
                    sizeof(uint16_t) * SEG_SIZE / M, 
                    hipMemcpyHostToDevice, 
                    stream[stream_index]) 
            )
        }

        // 2. Call kernels for every streams.
        for (int stream_index = 0; stream_index < STREAM_DIM; stream_index++) {
            int d_output_offset = stream_index * CEIL(SEG_SIZE, BLOCK_SIZE);
            int d_input_offset = stream_index * SEG_SIZE;
            pcrc16_kernel_reduction<<< SEG_SIZE / BLOCK_SIZE, BLOCK_SIZE, 0, stream[stream_index]>>>
                (d_message + d_input_offset / M, d_beta + d_input_offset / M, (uint32_t) generator + 0x10000, d_crc_partial_res + d_output_offset / M);
        }

        // 3. Copy outputs for every streams.
        for (int stream_index = 0; stream_index < STREAM_DIM; stream_index++) {
            int h_output_offset = ((i / SEG_SIZE) + stream_index) * CEIL(SEG_SIZE, BLOCK_SIZE);
            int d_output_offset = stream_index * CEIL(SEG_SIZE, BLOCK_SIZE);
            SAFE_CALL( 
                hipMemcpyAsync( 
                    h_crc_partial_res_dev + h_output_offset / M, 
                    d_crc_partial_res + d_output_offset / M, 
                    sizeof(uint16_t) * CEIL(SEG_SIZE, BLOCK_SIZE) / M, 
                    hipMemcpyDeviceToHost,
                    stream[stream_index]) )
        }
    }
    TM_device.stop();

    TM_host.start();
    for (size_t i = 0; i < (CEIL(N, BLOCK_SIZE) / M); i++) 
    {
        ((pcrc16_params_t *) params)->h_crc_res_dev ^= h_crc_partial_res_dev[i];
    }
    TM_host.stop();

    d_time->is_initialized      = true;
    d_time->is_task_parallelism = true;
    d_time->kernel_time = TM_device.duration() + TM_host.duration();
}

bool pcrc16_compare(const constants_t *c, void *params)
{
    uint16_t h_crc_res_dev  = ((pcrc16_params_t *) params)->h_crc_res_dev;
    uint16_t h_crc_res_host = ((pcrc16_params_t *) params)->h_crc_res_host;

    return h_crc_res_dev == h_crc_res_host;
}

bool pcrc16_compare_reduction(const constants_t *c, void *params)
{
    return pcrc16_compare(c, params);
}

bool pcrc16_compare_task_parallelism(const constants_t *c, void *params)
{
    return pcrc16_compare(c, params);
}

void pcrc16_free_common(void *params)
{
    uint8_t  *h_message    = ((pcrc16_params_t *) params)->h_message;
    uint16_t *h_message_16 = ((pcrc16_params_t *) params)->h_message_16;
    uint16_t *h_beta       = ((pcrc16_params_t *) params)->h_beta;
    uint16_t *h_crc_partial_res_dev = 
        ((pcrc16_params_t *) params)->h_crc_partial_res_dev;

    // Free host.
    delete[] h_message;
    delete[] h_message_16;
    delete[] h_beta;
    delete[] h_crc_partial_res_dev;
}

void pcrc16_free_device(void *params)
{
    uint16_t *d_message = ((pcrc16_params_t *) params)->d_message;
    uint16_t *d_beta    = ((pcrc16_params_t *) params)->d_beta; 
    uint16_t *d_crc_partial_res = ((pcrc16_params_t *) params)->d_crc_partial_res;

    // Free device.
    SAFE_CALL(hipFree(d_message))
    SAFE_CALL(hipFree(d_beta))
    SAFE_CALL(hipFree(d_crc_partial_res))
}

void pcrc16_free_device_reduction(void *params)
{
    pcrc16_free_device(params);
}

void pcrc16_free_device_task_parallelism(void *params)
{
    pcrc16_free_device(params);
}


void pcrc16_free(void *params)
{
    // Free host.
    pcrc16_free_common(params);
    // Free device.
    pcrc16_free_device(params);
    // Free params.
    delete ((pcrc16_params_t *) params);
}

void pcrc16_free_reduction(void *params) 
{
    // Free host.
    pcrc16_free_common(params);
    // Free device.
    pcrc16_free_device_reduction(params);
    // Free params.
    delete ((pcrc16_params_t *) params);
}

void pcrc16_free_task_parallelism(void *params) 
{
    // Free host.
    pcrc16_free_common(params);
    // Free device.
    pcrc16_free_device_task_parallelism(params);
    // Free params.
    delete ((pcrc16_params_t *) params);
}
//------------------------------------------------------------------------------

__global__
static void pcrc16_kernel(const uint16_t* d_message, 
                          const uint16_t* d_beta,
                          const uint32_t d_generator,
                          uint16_t *d_partial_crc)
{
    // __shared__ uint8_t ds_mem_message[BLOCK_SIZE];
    // __shared__ uint8_t ds_mem_beta[BLOCK_SIZE];
    __shared__ uint16_t ds_mem_crc[BLOCK_SIZE];
    uint32_t globalIndex = threadIdx.x + blockDim.x * blockIdx.x;

    // ds_mem_message[threadIdx.x] = d_message[globalIndex];
    // ds_mem_beta[threadIdx.x] = d_beta[globalIndex];
    uint16_t w    = d_message[globalIndex];
    uint16_t beta = d_beta[globalIndex];
    // __syncthreads();

    // Binary modulo 2 multiplication between w and beta.
    uint32_t mul = 0;
    for(uint8_t i = 0; i < (sizeof(uint16_t) * 8); i++)
    {
        if (beta & (1U << i))
        {
            mul ^= (uint32_t) w << i;
        }
    }

    uint32_t ret = 0;

    // Compute division of mul result by polynomial generator value.
    for (uint8_t i = 0; i < sizeof(uint32_t) * 8; i++)
    {
        // Compute subtraction.
        if ((ret & 0x00010000) != 0)
        {
            ret = (uint32_t)(ret ^ d_generator);
        }

        // Shift by 1 all the divident buffer.
        ret = (uint32_t)(ret << 1) 
            | (0x00000001 & (mul >> (sizeof(uint32_t) * 8 - i - 1)));
    }

    // Compute the last subtraction.
    if ((ret & 0x00010000) != 0)
    {
        ret = (uint32_t)(ret ^ d_generator);
    }

    ds_mem_crc[threadIdx.x] = (uint16_t) ret;
    __syncthreads();

    if (threadIdx.x == 0) 
    {
        uint16_t partial_crc = 0;
        for (size_t i = 0; i < BLOCK_SIZE; i++)
        {
            partial_crc ^= ds_mem_crc[i];
        }
        d_partial_crc[blockIdx.x] = partial_crc;
    }
}

__global__
static void pcrc16_kernel_reduction(const uint16_t* d_message, 
                                    const uint16_t* d_beta,
                                    const uint32_t d_generator,
                                    uint16_t *d_partial_crc)
{
    // __shared__ uint8_t ds_mem_message[BLOCK_SIZE];
    // __shared__ uint8_t ds_mem_beta[BLOCK_SIZE];
    __shared__ uint16_t ds_mem_crc[BLOCK_SIZE];
    uint32_t globalIndex = threadIdx.x + blockDim.x * blockIdx.x;

    // ds_mem_message[threadIdx.x] = d_message[globalIndex];
    // ds_mem_beta[threadIdx.x] = d_beta[globalIndex];
    uint16_t w    = d_message[globalIndex];
    uint16_t beta = d_beta[globalIndex];
    // __syncthreads();

    // Binary modulo 2 multiplication between w and beta.
    uint32_t mul = 0;
    for(uint8_t i = 0; i < (sizeof(uint16_t) * 8); i++)
    {
        if (beta & (1U << i))
        {
            mul ^= (uint32_t) w << i;
        }
    }

    uint32_t ret = 0;

    // Compute division of mul result by polynomial generator value.
    for (uint8_t i = 0; i < sizeof(uint32_t) * 8; i++)
    {
        // Compute subtraction.
        if ((ret & 0x00010000) != 0)
        {
            ret = (uint32_t)(ret ^ d_generator);
        }

        // Shift by 1 all the divident buffer.
        ret = (uint32_t)(ret << 1) 
            | (0x00000001 & (mul >> (sizeof(uint32_t) * 8 - i - 1)));
    }

    // Compute the last subtraction.
    if ((ret & 0x00010000) != 0)
    {
        ret = (uint32_t)(ret ^ d_generator);
    }

    ds_mem_crc[threadIdx.x] = (uint16_t) ret;
    __syncthreads();

    // Perform Reduction.
    for (size_t i = 1; i < blockDim.x; i *= 2) {
        size_t index = threadIdx.x * i * 2;
        if (index < blockDim.x) {
            ds_mem_crc[index] ^= ds_mem_crc[index + i]; 
        }

        __syncthreads();
    }

    // Write back in memory.
    if (threadIdx.x == 0) {
        d_partial_crc[blockIdx.x] = ds_mem_crc[0];
    }
}
